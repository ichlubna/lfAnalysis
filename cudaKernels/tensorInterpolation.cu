#include "hip/hip_runtime.h"
__device__ bool coordsOutside(uint2 coords)
{
    constexpr unsigned int PX_PER_WARP{8};
    if(coords.x >= IMG_WIDTH*PX_PER_WARP || coords.y >= IMG_HEIGHT)
        return false;
}

__device__ void interpolateImages(Images images, half weights[WEIGHTS_ROWS][WEIGHTS_COLS], uint2 coords, int focus)
{
    constexpr int MAT_PX_COUNT{8};
    constexpr int WARP_COUNT{8}; 
    constexpr int OUT_VIEWS_COUNT{WEIGHTS_COLS}; 
    constexpr int MAT_VIEW_COUNT{16};
    constexpr int MATS_PER_WARP{1};

    int warpID = threadIdx.x/WARP_SIZE;
    uint2 pxCoords{coords.x/CHANNEL_COUNT, coords.y};
    int channelID = threadIdx.x%CHANNEL_COUNT;
    //int matrixRowID = CHANNELS*((int)(coords.x%WARP_SIZE)/CHANNELS) + channelID;
    int matrixRowID = threadIdx.x%WARP_SIZE;//coords.x%WARP_SIZE;
    float2 gridCenter{(GRID_COLS-1)/2.f, (GRID_ROWS-1)/2.f};

    extern __shared__ half localMemory[];
    MemoryPartitioner memoryPartitioner(localMemory);
   
    auto pixelMatrix = memoryPartitioner.getMatrix(MATS_PER_WARP*WARP_COUNT, MAT_PX_COUNT*CHANNEL_COUNT, MAT_VIEW_COUNT); 
    auto resultMatrix = memoryPartitioner.getMatrix(MATS_PER_WARP*WARP_COUNT, MAT_PX_COUNT*CHANNEL_COUNT, OUT_VIEWS_COUNT);
    auto localWeights = memoryPartitioner.getMatrix(1, WEIGHTS_ROWS, WEIGHTS_COLS);
    loadWeightsSync(weights[0], localWeights.data);  

    wmma::fragment<wmma::accumulator, 32, 8, 16, half> matResult;
    wmma::fill_fragment(matResult, 0.0f);
    wmma::fragment<wmma::matrix_a, 32, 8, 16, half, wmma::row_major> matPixels;
    wmma::fragment<wmma::matrix_b, 32, 8, 16, half, wmma::row_major> matWeights;
    
    int batchCount = (GRID_COLS*GRID_ROWS)/MAT_VIEW_COUNT;
    for(int i=0; i<batchCount; i++)
    {
        wmma::load_matrix_sync(matWeights, localWeights.ptr(0, i*MAT_VIEW_COUNT, 0), localWeights.stride());

        for(int j=0; j<MAT_VIEW_COUNT; j+=2)
        {
            int gridID = i*MAT_VIEW_COUNT+j; 
            int2 focusedCoords = focusCoords(pxCoords, 10, {(unsigned int)gridID/GRID_COLS, (unsigned int)gridID%GRID_COLS}, gridCenter);
            auto pixel = images.getPixelAsArray<half>(gridID, focusedCoords);
            half2 channelPair{pixel[channelID],0};
            gridID++;
            focusedCoords = focusCoords(pxCoords, 10, {(unsigned int)gridID/GRID_COLS, (unsigned int)gridID%GRID_COLS}, gridCenter);
            pixel = images.getPixelAsArray<half>(gridID, focusedCoords);
            channelPair.y = pixel[channelID];
            pixelMatrix.ref<half2>(warpID, matrixRowID, j) = channelPair;
        }
        wmma::load_matrix_sync(matPixels, pixelMatrix.ptr(warpID, 0, 0), pixelMatrix.stride());
        wmma::mma_sync(matResult, matPixels, matWeights, matResult);
    }
    
    wmma::store_matrix_sync(resultMatrix.ptr(warpID, 0, 0), matResult, OUT_VIEWS_COUNT, wmma::mem_row_major);
  
    for(int i=0; i<OUT_VIEWS_COUNT; i++) 
        images.setChannel(i, pxCoords, channelID, round((float)resultMatrix.ref(warpID, matrixRowID, i)));
}
